#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"

// represents the objects in the system.  Global variables
vector3 *hVel, *d_hVel;
vector3 *hPos, *d_hPos;
vector3 **d_accels, *d_accel_sum;
vector3* temp[NUMENTITIES];
double *mass, *d_mass;

//initHostMemory: Create storage for numObjects entities in our system
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

//initDeviceMemory: Create storage for numObjects entities in our system on the GPU
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the for the device variables on the GPU

//I was going to do this in a function, but it causes errors on Darwin (and only on darwin?)
/*
void initDeviceMemory() {
	hipMalloc((void**)&d_accels, NUMENTITIES * sizeof(vector3*));

	//2d arrays are funky in cuda, here's a workaround
	for (int i = 0; i < NUMENTITIES; i++) {
		hipMalloc(&temp[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(d_accels, temp, NUMENTITIES * sizeof(vector3*), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_hPos, NUMENTITIES*sizeof(vector3));

	hipMalloc((void**)&d_hVel, NUMENTITIES*sizeof(vector3));

	hipMalloc((void**)&d_accel_sum, NUMENTITIES*sizeof(vector3));

	hipMalloc((void**)&d_mass, NUMENTITIES*sizeof(double));
}
*/

//freeDeviceMemory: Free storage allocated by a previous call to initDeviceMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to device global variables
void freeDeviceMemory() {
	for (int i = 0; i < NUMENTITIES; i++) {
		hipFree(temp[i]);
	}

	hipFree(d_accels);

	hipFree(d_hPos);

	hipFree(d_hVel);

	hipFree(d_accel_sum);
	
	hipFree(d_mass);
}


//freeHostMemory: Free storage allocated by a previous call to initHostMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
}

//planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
//Parameters: None
//Returns: None
//Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

//randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
//Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
//Returns: None
//Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j= start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

//printSystem: Prints out the entire system to the supplied file
//Parameters: 	handle: A handle to an open file with write access to prnt the data to
//Returns: 		none
//Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		printf("),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		fprintf(handle,"),m=%lf\n",mass[i]);
	}
}

int main(int argc, char **argv)
{
	clock_t t0=clock();
	int t_now;
	//srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);

	//The cuda malloc statements are here instead of the function because darwin breaks if its in a function :))))))))))

	hipMalloc((void**)&d_accels, NUMENTITIES * sizeof(vector3*));

	//2d arrays are funky in cuda, here's a workaround
	for (int i = 0; i < NUMENTITIES; i++) {
		hipMalloc(&temp[i], sizeof(vector3) * NUMENTITIES);
	}
	hipMemcpy(d_accels, temp, NUMENTITIES * sizeof(vector3*), hipMemcpyHostToDevice);

	hipMalloc((void**)&d_hPos, NUMENTITIES*sizeof(vector3));

	hipMalloc((void**)&d_hVel, NUMENTITIES*sizeof(vector3));

	hipMalloc((void**)&d_accel_sum, NUMENTITIES*sizeof(vector3));

	hipMalloc((void**)&d_mass, NUMENTITIES*sizeof(double));

	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);
	//now we have a system.
	#ifdef DEBUG
	printSystem(stdout);
	#endif

	//we need to send the system to the GPU
	hipMemcpy(d_hPos, hPos, NUMENTITIES*sizeof(vector3), hipMemcpyHostToDevice);
	hipMemcpy(d_hVel, hVel, NUMENTITIES*sizeof(vector3), hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, NUMENTITIES*sizeof(double), hipMemcpyHostToDevice);

	for (t_now=0;t_now<DURATION;t_now+=INTERVAL){
		compute();
	}

	hipMemcpy(hPos, d_hPos, NUMENTITIES*sizeof(vector3), hipMemcpyDeviceToHost);

	hipMemcpy(hVel, d_hVel, NUMENTITIES*sizeof(vector3), hipMemcpyDeviceToHost);

	clock_t t1=clock()-t0;
#ifdef DEBUG
	printSystem(stdout);
#endif
	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
	freeDeviceMemory();
}
